#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if( i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    printf("hello cuda. vector add\n");
    int numElementCount = 50000;
    int byteCount = numElementCount * sizeof(float);
    printf("malloc for A, B, C. size=%d\n", byteCount);
    float* pA = (float*)malloc(byteCount);
    float* pB = (float*)malloc(byteCount);
    float* pC = (float*)malloc(byteCount);
    for(int i=0; i<numElementCount; i++)
    {
        pA[i] = rand() / (float)RAND_MAX;
        pB[i] = rand() / (float)RAND_MAX;
    }

    printf("hipMalloc for A, B, C. size=%d\n", byteCount);
    float* pA_d = NULL;
    hipError_t err = hipMalloc((void**)&pA_d, byteCount);
    if(err!=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for A. error code %d, error: %s\n",
                err, hipGetErrorString(err));
        return -1;
    }
    float* pB_d = nullptr;
    err = hipMalloc((void**)&pB_d, byteCount);
    if(err!=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for B. error code %d, error: %s\n",
                err, hipGetErrorString(err));
        return -1;
    }
    float* pC_d = nullptr;
    err = hipMalloc((void**)&pC_d, byteCount);
    if(err!=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for C. error code %d, error: %s\n",
                err, hipGetErrorString(err));
        return -1;
    }

    printf("copy A, B vector data from host memory to device memory\n");
    err = hipMemcpy(pA_d, pA, byteCount, hipMemcpyHostToDevice);
    if(err!=hipSuccess)
    {
        fprintf(stderr, "Failed to copy A from host memory to device memory. error code=%d msg:%s\n",
                err, hipGetErrorString(err));
        return -2;
    }
    err = hipMemcpy(pB_d, pB, byteCount, hipMemcpyHostToDevice);
    if(err!=hipSuccess)
    {
        fprintf(stderr, "Failed to copy B from host memory to device memory. error code=%d msg:%s\n",
                err, hipGetErrorString(err));
        return -2;
    }

// Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElementCount + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
            threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(pA_d, pB_d, pC_d, numElementCount);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
    {
        printf("do vectorAdd success.err=%d\n", err);
    }
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(pC, pC_d, byteCount, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElementCount; ++i) {
        if (fabs(pA[i] + pB[i] - pC[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("test success\n");

    hipFree(pA_d);
    hipFree(pB_d);
    hipFree(pC_d);
    free(pA);
    free(pB);
    free(pC);
    return 0;
}
